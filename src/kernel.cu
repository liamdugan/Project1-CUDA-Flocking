#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

thrust::device_ptr<int> dev_thrust_pos;
thrust::device_ptr<int> dev_thrust_vel1;
thrust::device_ptr<int> dev_thrust_vel2;

glm::vec3 *dev_shuffle_pos;
glm::vec3 *dev_shuffle_vel1;
glm::vec3 *dev_shuffle_vel2;

thrust::device_ptr<int> dev_thrust_shuffle_pos;
thrust::device_ptr<int> dev_thrust_shuffle_vel1;
thrust::device_ptr<int> dev_thrust_shuffle_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?

thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  hipMalloc((void**)&dev_shuffle_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_shuffle_pos failed!");

  hipMalloc((void**)&dev_shuffle_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_shuffle_vel1 failed!");

  hipMalloc((void**)&dev_shuffle_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_shuffle_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");

  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

  dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices);
  dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {

  // Initialize the vectors that will be edited by the three rules
  glm::vec3 rule1_vel(0.0f, 0.0f, 0.0f);
  glm::vec3 rule2_vel(0.0f, 0.0f, 0.0f);
  glm::vec3 rule3_vel(0.0f, 0.0f, 0.0f);

  // Initialize other state relating to the rules
  glm::vec3 rule1_perceived_center(0.0f, 0.0f, 0.0f);
  glm::vec3 rule2_c_vel(0.0f, 0.0f, 0.0f);
  glm::vec3 rule3_perceived_vel(0.0f, 0.0f, 0.0f);
  
  // Loop through all of the boids
  int rule1Count = 0;
  int rule3Count = 0;
  for (int i = 0; i < N; ++i)
  {
    if (i != iSelf)
    {
      glm::vec3 difference = pos[i] - pos[iSelf];
      float len = glm::length(difference);

      // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
      if (len < rule1Distance)
      {
        rule1Count++;
        rule1_perceived_center += pos[i];
      }
      
      // Rule 2: boids try to stay a distance d away from each other
      if (len < rule2Distance)
      {
        rule2_c_vel -= (pos[i] - pos[iSelf]);
      }

      // Rule 3: boids try to match the speed of surrounding boids
      if (len < rule3Distance)
      {
        rule3Count++;
        if (glm::length(vel[i]) > 0)
        {
          rule3_perceived_vel += vel[i];
        }
      }
    }
  }

  // Scale the perceived center by the number of Boids
  if (rule1Count > 0)
  {
    rule1_perceived_center /= rule1Count;
    rule1_vel = (rule1_perceived_center - pos[iSelf]) * rule1Scale;
  }

  if (rule3Count > 0)
  {
    rule3_perceived_vel /= rule3Count;
    rule3_vel = rule3_perceived_vel * rule3Scale;
  }

  // Calculate the velocities from the rules
  rule2_vel = rule2_c_vel * rule2Scale;

  return vel[iSelf] + rule1_vel + rule2_vel + rule3_vel;
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  
  // Compute a new velocity based on pos and vel1
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }

  glm::vec3 newVelocity = computeVelocityChange(N, index, pos, vel1);

  // Clamp the speed (?)
//  newVelocity = glm::clamp(newVelocity, -maxSpeed, maxSpeed);
  if (glm::length(newVelocity) > maxSpeed)
  {
    newVelocity = (newVelocity / glm::length(newVelocity)) * maxSpeed;
  }

  // Record the new velocity into vel2. Question: why NOT vel1?
  vel2[index] = newVelocity;
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {

  // we're assuming that this is only called once at the very beginning (???)

  // get the index of dis boid
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  
  // get my position relative to the minimum corner of the grid
  glm::vec3 myPos = pos[index];
  glm::vec3 gridLocation = myPos - gridMin;
  float myPosX = gridLocation.x;
  float myPosY = gridLocation.y;
  float myPosZ = gridLocation.z;
  
  // get my grid cell in 1D index
  int gridCellX = static_cast<int>(myPosX * inverseCellWidth);
  int gridCellY = static_cast<int>(myPosY * inverseCellWidth);
  int gridCellZ = static_cast<int>(myPosZ * inverseCellWidth);
  int gridIndex = gridIndex3Dto1D(gridCellX, gridCellY, gridCellZ, gridResolution);

  // - Label the boid with the index of its grid cell.
  gridIndices[index] = gridIndex;

  // - Set up a parallel array of integer indices as pointers to the actual
  // boid data in pos and vel1/vel2
  indices[index] = index;
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"

  // get the index of the thread
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }

  // now just check that index, and the indices to the left and right
  int myGridIndex = particleGridIndices[index];

  // if the index to the left is different, or if there isn't an index to the left
  // then mark this as the start of grid myGridIndex
  if (index == 0 || (index > 0 && myGridIndex != particleGridIndices[index - 1]))
  {
    gridCellStartIndices[myGridIndex] = index;
  }

  // if the index to the right is different, or if there isn't an index to the right 
  // then mark this as the end of grid myGridIndex
  if (index == (N-1) || (index < (N - 1) && myGridIndex != particleGridIndices[index + 1]))
  {
    gridCellEndIndices[myGridIndex] = index;
  }
}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.

  // first, as per usual, get my index
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }

  // just like the naive version start by initializing this stuff
  // Initialize the vectors that will be edited by the three rules
  glm::vec3 rule1_vel(0.0f, 0.0f, 0.0f);
  glm::vec3 rule2_vel(0.0f, 0.0f, 0.0f);
  glm::vec3 rule3_vel(0.0f, 0.0f, 0.0f);

  // Initialize other state relating to the rules
  glm::vec3 rule1_perceived_center(0.0f, 0.0f, 0.0f);
  glm::vec3 rule2_c_vel(0.0f, 0.0f, 0.0f);
  glm::vec3 rule3_perceived_vel(0.0f, 0.0f, 0.0f);
  
  // - Get the grid cell that this particle is in using the old fashioned way
  // start by getting my position relative to the minimum corner of the grid
  glm::vec3 myPos = pos[particleArrayIndices[index]];
  glm::vec3 gridLocation = myPos - gridMin;
  float myPosX = gridLocation.x;
  float myPosY = gridLocation.y;
  float myPosZ = gridLocation.z;
  
  // get my grid cell in 3D float index
  float gridCellX = myPosX * inverseCellWidth;
  float gridCellY = myPosY * inverseCellWidth;
  float gridCellZ = myPosZ * inverseCellWidth;

  // also get my grid cell in int index
  int gridCellXIndex = static_cast<int>(gridCellX);
  int gridCellYIndex = static_cast<int>(gridCellY);
  int gridCellZIndex = static_cast<int>(gridCellZ);

  // also get the middle of each one of those grid cells
  float gridCellXMiddle = static_cast<float>(static_cast<int>(gridCellX)) + 0.5f;
  float gridCellYMiddle = static_cast<float>(static_cast<int>(gridCellY)) + 0.5f;
  float gridCellZMiddle = static_cast<float>(static_cast<int>(gridCellZ)) + 0.5f;
  
  // now determine which octant we are in by the three numbers
  int xSign = (gridCellX < gridCellXMiddle) ? -1 : 1;
  int ySign = (gridCellY < gridCellYMiddle) ? -1 : 1;
  int zSign = (gridCellZ < gridCellZMiddle) ? -1 : 1;

  // for loop over the 8 neighbors to check for neighboring boids
  int rule1Count = 0;
  int rule3Count = 0;
  for (int i = 0; i < 8; ++i)
  {
    // first we have to figure out the 3D grid index of the neighbor

    // initialize to our index
    int neighborGridX = gridCellXIndex;
    int neighborGridY = gridCellYIndex;
    int neighborGridZ = gridCellZIndex;

    // based on the three numbers from earlier, calculate the true neighbor grid index
    if (i % 2 > 0)
    {
      neighborGridX += xSign;
    }

    if (i % 4 > 1)
    {
      neighborGridY += ySign;
    }
    
    if (i % 8 > 3)
    {
      neighborGridZ += zSign;
    }

    // if the neighbor's grid index is invalid, then continue
    if (neighborGridX < 0 || neighborGridX > (gridResolution - 1) ||
      neighborGridY < 0 || neighborGridY > (gridResolution - 1) ||
      neighborGridZ < 0 || neighborGridZ > (gridResolution - 1))
    {
      continue;
    }
    
    // now that we have the real neighbor grid, get that in 1D and find out where it starts and ends
    int neighborGridIndex = gridIndex3Dto1D(neighborGridX, neighborGridY, neighborGridZ, gridResolution);
    int neighborGridParticleStart = gridCellStartIndices[neighborGridIndex];
    int neighborGridParticleEnd = gridCellEndIndices[neighborGridIndex];

    // if the grids do not have any neighbors or if there is an error then that's fine, just continue
    if (neighborGridParticleStart == -1 || neighborGridParticleEnd == -1 || neighborGridParticleStart > neighborGridParticleEnd)
    {
      continue;
    }
 
    // for each neighbor now use that velocity to update our boid here
    for (int j = neighborGridParticleStart; j <= neighborGridParticleEnd; ++j)
    {
      if (j != index)
      {
        glm::vec3 difference = pos[particleArrayIndices[j]] - pos[particleArrayIndices[index]];
        float len = glm::length(difference);
     
        // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
        if (len < rule1Distance)
        {
          rule1Count++;
          rule1_perceived_center += pos[particleArrayIndices[j]];
        }

        // Rule 2: boids try to stay a distance d away from each other
        if (len < rule2Distance)
        {
          rule2_c_vel -= (pos[particleArrayIndices[j]] - pos[particleArrayIndices[index]]);
        }

        // Rule 3: boids try to match the speed of surrounding boids
        if (len < rule3Distance)
        {
          rule3Count++;
          if (glm::length(vel1[particleArrayIndices[j]]) > 0)
          {
            rule3_perceived_vel += vel1[particleArrayIndices[j]];
          }
        }
      }
    }
  }

  // Scale rules 1 and 3 by the number of Neighbors
  if (rule1Count > 0)
  {
    rule1_perceived_center /= rule1Count;
    rule1_vel = (rule1_perceived_center - pos[particleArrayIndices[index]]) * rule1Scale;
  }

  if (rule3Count > 0)
  {
    rule3_perceived_vel /= rule3Count;
    rule3_vel = rule3_perceived_vel * rule3Scale;
  }

  // Rule 2 doesn't need to be scaled
  rule2_vel = rule2_c_vel * rule2Scale;

  // calculate the new velocity by adding up all the rule results
  glm::vec3 newVelocity = vel1[particleArrayIndices[index]] + rule1_vel + rule2_vel + rule3_vel;
  
  // scale the velocity to account for max speed
  if (glm::length(newVelocity) > maxSpeed)
  {
    newVelocity = (newVelocity / glm::length(newVelocity)) * maxSpeed;
  }

  // put it into vel2
  vel2[particleArrayIndices[index]] = newVelocity;
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2
  // first, as per usual, get my index
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }

  // just like the naive version start by initializing this stuff
  // Initialize the vectors that will be edited by the three rules
  glm::vec3 rule1_vel(0.0f, 0.0f, 0.0f);
  glm::vec3 rule2_vel(0.0f, 0.0f, 0.0f);
  glm::vec3 rule3_vel(0.0f, 0.0f, 0.0f);

  // Initialize other state relating to the rules
  glm::vec3 rule1_perceived_center(0.0f, 0.0f, 0.0f);
  glm::vec3 rule2_c_vel(0.0f, 0.0f, 0.0f);
  glm::vec3 rule3_perceived_vel(0.0f, 0.0f, 0.0f);
  
  // - Get the grid cell that this particle is in using the old fashioned way
  // start by getting my position relative to the minimum corner of the grid
  glm::vec3 myPos = pos[index];
  glm::vec3 gridLocation = myPos - gridMin;
  float myPosX = gridLocation.x;
  float myPosY = gridLocation.y;
  float myPosZ = gridLocation.z;
  
  // get my grid cell in 3D float index
  float gridCellX = myPosX * inverseCellWidth;
  float gridCellY = myPosY * inverseCellWidth;
  float gridCellZ = myPosZ * inverseCellWidth;

  // also get my grid cell in int index
  int gridCellXIndex = static_cast<int>(gridCellX);
  int gridCellYIndex = static_cast<int>(gridCellY);
  int gridCellZIndex = static_cast<int>(gridCellZ);

  // also get the middle of each one of those grid cells
  float gridCellXMiddle = static_cast<float>(static_cast<int>(gridCellX)) + 0.5f;
  float gridCellYMiddle = static_cast<float>(static_cast<int>(gridCellY)) + 0.5f;
  float gridCellZMiddle = static_cast<float>(static_cast<int>(gridCellZ)) + 0.5f;
  
  // now determine which octant we are in by the three numbers
  int xSign = (gridCellX < gridCellXMiddle) ? -1 : 1;
  int ySign = (gridCellY < gridCellYMiddle) ? -1 : 1;
  int zSign = (gridCellZ < gridCellZMiddle) ? -1 : 1;

  // for loop over the 8 neighbors to check for neighboring boids
  int rule1Count = 0;
  int rule3Count = 0;
  for (int i = 0; i < 8; ++i)
  {
    // first we have to figure out the 3D grid index of the neighbor

    // initialize to our index
    int neighborGridX = gridCellXIndex;
    int neighborGridY = gridCellYIndex;
    int neighborGridZ = gridCellZIndex;

    // based on the three numbers from earlier, calculate the true neighbor grid index
    if (i % 2 > 0)
    {
      neighborGridX += xSign;
    }

    if (i % 4 > 1)
    {
      neighborGridY += ySign;
    }
    
    if (i % 8 > 3)
    {
      neighborGridZ += zSign;
    }

    // if the neighbor's grid index is invalid, then continue
    if (neighborGridX < 0 || neighborGridX > (gridResolution - 1) ||
      neighborGridY < 0 || neighborGridY > (gridResolution - 1) ||
      neighborGridZ < 0 || neighborGridZ > (gridResolution - 1))
    {
      continue;
    }
    
    // now that we have the real neighbor grid, get that in 1D and find out where it starts and ends
    int neighborGridIndex = gridIndex3Dto1D(neighborGridX, neighborGridY, neighborGridZ, gridResolution);
    int neighborGridParticleStart = gridCellStartIndices[neighborGridIndex];
    int neighborGridParticleEnd = gridCellEndIndices[neighborGridIndex];

    // if the grids do not have any neighbors or if there is an error then that's fine, just continue
    if (neighborGridParticleStart == -1 || neighborGridParticleEnd == -1 || neighborGridParticleStart > neighborGridParticleEnd)
    {
      continue;
    }
 
    // for each neighbor now use that velocity to update our boid here
    for (int j = neighborGridParticleStart; j <= neighborGridParticleEnd; ++j)
    {
      if (j != index)
      {
        glm::vec3 difference = pos[j] - pos[index];
        float len = glm::length(difference);
     
        // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
        if (len < rule1Distance)
        {
          rule1Count++;
          rule1_perceived_center += pos[j];
        }

        // Rule 2: boids try to stay a distance d away from each other
        if (len < rule2Distance)
        {
          rule2_c_vel -= (pos[j] - pos[index]);
        }

        // Rule 3: boids try to match the speed of surrounding boids
        if (len < rule3Distance)
        {
          rule3Count++;
          if (glm::length(vel1[j]) > 0)
          {
            rule3_perceived_vel += vel1[j];
          }
        }
      }
    }
  }

  // Scale rules 1 and 3 by the number of Neighbors
  if (rule1Count > 0)
  {
    rule1_perceived_center /= rule1Count;
    rule1_vel = (rule1_perceived_center - pos[index]) * rule1Scale;
  }

  if (rule3Count > 0)
  {
    rule3_perceived_vel /= rule3Count;
    rule3_vel = rule3_perceived_vel * rule3Scale;
  }

  // Rule 2 doesn't need to be scaled
  rule2_vel = rule2_c_vel * rule2Scale;

  // calculate the new velocity by adding up all the rule results
  glm::vec3 newVelocity = vel1[index] + rule1_vel + rule2_vel + rule3_vel;
  
  // scale the velocity to account for max speed
  if (glm::length(newVelocity) > maxSpeed)
  {
    newVelocity = (newVelocity / glm::length(newVelocity)) * maxSpeed;
  }

  // put it into vel2
  vel2[index] = newVelocity;
}

__global__ void kernShuffle(int N, int* particleArrayIndices, glm::vec3* pos,
  glm::vec3* pos_shuffle, glm::vec3* vel1, glm::vec3* vel1_shuffle,
  glm::vec3* vel2, glm::vec3* vel2_shuffle) {

  // get the index of the thing to switch
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }

  // shuffle all of the indices in parallel
  pos_shuffle[index] = pos[particleArrayIndices[index]];
  vel1_shuffle[index] = vel1[particleArrayIndices[index]];
  vel2_shuffle[index] = vel2[particleArrayIndices[index]];
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
  kernUpdateVelocityBruteForce << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_pos, dev_vel1, dev_vel2);
  kernUpdatePos << <fullBlocksPerGrid, blockSize >> > (numObjects, dt, dev_pos, dev_vel1);

  // TODO-1.2 ping-pong the velocity buffers
  glm::vec3 *temp = dev_vel1;
  dev_vel1 = dev_vel2;
  dev_vel2 = temp;
}

void Boids::stepSimulationScatteredGrid(float dt) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
  kernComputeIndices << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum,
    gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
  checkCUDAErrorWithLine("kernComputeIndices failed!");
    hipDeviceSynchronize();
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + (numObjects - 1), dev_thrust_particleArrayIndices);
  checkCUDAErrorWithLine("thrust sort_by_key failed!");
    hipDeviceSynchronize();
  // reset the grid start and end indices back to the default of -1
  kernResetIntBuffer << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_gridCellStartIndices, -1);
  checkCUDAErrorWithLine("kernResetIntBuffer failed!");
    hipDeviceSynchronize();
  kernResetIntBuffer << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_gridCellEndIndices, -1);
  checkCUDAErrorWithLine("kernResetIntBuffer failed!");
    hipDeviceSynchronize();
    // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  kernIdentifyCellStartEnd << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_particleGridIndices,
    dev_gridCellStartIndices, dev_gridCellEndIndices);
  checkCUDAErrorWithLine("kernIdentifyCellStartEnd failed!");
  hipDeviceSynchronize();  
  // - Perform velocity updates using neighbor search
  kernUpdateVelNeighborSearchScattered << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount,
    gridMinimum, gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, dev_particleArrayIndices,
    dev_pos, dev_vel1, dev_vel2);
  checkCUDAErrorWithLine("kernUpdateVelNeighborSearchScattered failed!");
 
  hipDeviceSynchronize();

  // - Update positions
  kernUpdatePos << <fullBlocksPerGrid, blockSize >> > (numObjects, dt, dev_pos, dev_vel1);
  checkCUDAErrorWithLine("kernUpdatePos failed!");
   // hipDeviceSynchronize();
  // - Ping-pong buffers as needed
  glm::vec3 *temp = dev_vel1;
  dev_vel1 = dev_vel2;
  dev_vel2 = temp;
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.

  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
  kernComputeIndices << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum,
    gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
  checkCUDAErrorWithLine("kernComputeIndices failed!");
    hipDeviceSynchronize();
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + (numObjects - 1), dev_thrust_particleArrayIndices);
  checkCUDAErrorWithLine("thrust sort_by_key failed!");

  // reset the grid start and end indices back to the default of -1
  kernResetIntBuffer << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_gridCellStartIndices, -1);
  checkCUDAErrorWithLine("kernResetIntBuffer failed!");

  kernResetIntBuffer << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_gridCellEndIndices, -1);
  checkCUDAErrorWithLine("kernResetIntBuffer failed!");

  hipDeviceSynchronize();

  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  kernIdentifyCellStartEnd << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_particleGridIndices,
    dev_gridCellStartIndices, dev_gridCellEndIndices);
  checkCUDAErrorWithLine("kernIdentifyCellStartEnd failed!");
  
  // reshuffle all of the particle data
  kernShuffle << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_particleArrayIndices, dev_pos, 
    dev_shuffle_pos, dev_vel1, dev_shuffle_vel1, dev_vel2, dev_shuffle_vel2);
  checkCUDAErrorWithLine("kernShuffle failed!");

  hipDeviceSynchronize();

  // - Perform velocity updates using neighbor search
  kernUpdateVelNeighborSearchCoherent << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount,
    gridMinimum, gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices,
    dev_shuffle_pos, dev_shuffle_vel1, dev_shuffle_vel2);
  checkCUDAErrorWithLine("kernUpdateVelNeighborSearchScattered failed!");
 
  hipDeviceSynchronize();

  // - Update positions
  kernUpdatePos << <fullBlocksPerGrid, blockSize >> > (numObjects, dt, dev_shuffle_pos, dev_shuffle_vel1);
  checkCUDAErrorWithLine("kernUpdatePos failed!");


  // ping-pong the newly shuffled positions
  glm::vec3 *pingPongTemp1 = dev_pos;
  glm::vec3 *pingPongTemp2 = dev_vel1;
  glm::vec3 *pingPongTemp3 = dev_vel2;
  dev_pos = dev_shuffle_pos;
  dev_vel1 = dev_shuffle_vel1;
  dev_vel2 = dev_shuffle_vel2;
  dev_shuffle_pos = pingPongTemp1;
  dev_shuffle_vel1 = pingPongTemp2;
  dev_shuffle_vel2 = pingPongTemp3;

  // ping pong vel1 and vel2
  glm::vec3 *temp = dev_vel1;
  dev_vel1 = dev_vel2;
  dev_vel2 = temp;
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);
  hipFree(dev_shuffle_vel1);
  hipFree(dev_shuffle_vel2);
  hipFree(dev_shuffle_pos);
  

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]>intKeys{ new int[N] };
  std::unique_ptr<int[]>intValues{ new int[N] };

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
